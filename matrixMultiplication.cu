
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

__global__
void multiply(int *a, int *b, int *c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    c[row * N + col] = 0;
    for (size_t i = 0; i < N; i++)
    {
        c[row * N + col] += a[row * N + i] * b[i * N + col];;
    }
    
}

void verify_result(int *a, int *b, int *c, int N) {
  // For every row...
  for (int i = 0; i < N; i++) {
    // For every column...
    for (int j = 0; j < N; j++) {
      // For every element in the row-column pair
      int tmp = 0;
      for (int k = 0; k < N; k++) {
        // Accumulate the partial results
        tmp += a[i * N + k] * b[k * N + j];
      }

      // Check against the CPU result
      assert(tmp == c[i * N + j]);
    }
  }
}

void init_matrices(int *a, int *b, int N) {
  for (size_t i = 0; i < N; i++) {
    for (size_t j = 0; j < N; j++) {
        a[i * N + j] = rand() % 100;
        b[i * N + j] = rand() % 100;
    }
    
  }
  
}

int main(int argc, char const *argv[])
{
  constexpr int N = 1 << 10;

  constexpr size_t bytes = N * N * sizeof(int);

  int *h_a, *h_b, *h_c;

  h_a = (int*)malloc(bytes);
  h_b = (int*)malloc(bytes);
  h_c = (int*)malloc(bytes);

  int *d_a, *d_b, *d_c;

  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);

  init_matrices(h_a, h_b, N);

  hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

  int BLOCK_SIZE = 16;

  int GRID_SIZE = (int) ceil(N / BLOCK_SIZE);

  dim3 grid(GRID_SIZE, GRID_SIZE);
  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

  multiply <<<grid, threads>>> (d_a, d_b, d_c, N);


  hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);

  std::cout << "Finish" << std::endl;

  verify_result(h_a, h_b, h_c, N);
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);

  return 0;
}
